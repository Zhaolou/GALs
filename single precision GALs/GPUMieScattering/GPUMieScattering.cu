#include "hip/hip_runtime.h"
// GPUMieScattering.cpp : �������̨Ӧ�ó������ڵ㡣
//
#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <helper_functions.h>
#include <hip/device_functions.h>
#include <time.h>
__global__ void kernel2(int n, float *g_data)
{
	printf("threadID is %d", threadIdx.x);
    const unsigned int tid = threadIdx.x;
    int data = g_data[tid];
	if(tid < n)
    g_data[tid] = g_data[tid] + 1;
}


//__global__ void kernel(float mr, float mi, float x, float *u_data, int N)
__global__ void kernel(float mr, float mi, float x, float *u_data, int N, int nmax, float* abr_data, 
	float* abi_data,float* s1r_data, float* s1i_data, float* s2r_data, float* s2i_data, float* p, float *t)
{

    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
    float u = u_data[tid];
	if(tid < N)
	{
		p[0 + tid * nmax] = 1; t[0 + tid * nmax] = u;
		p[1 + tid * nmax] = 3 * u; t[1 + tid * nmax] = 3 * cos(2 * acos(u));
		float p1, p2, t1, t2;
		int n1;
		for (n1 = 3; n1 <= nmax; n1++)
		{
			p1 = (2 * n1 - 1.0) / (n1 - 1) * p[n1 - 2 + tid * nmax] * u;
			p2 = n1 * 1.0 / (n1 - 1) * p[n1 - 3 + tid * nmax];
			p[n1 - 1 + tid * nmax] = p1 - p2;
			t1 = n1 * u * p[n1 - 1 + tid * nmax];
			t2 = (n1 + 1) * p[n1 - 2 + tid * nmax];
			t[n1 - 1 + tid * nmax] = t1 - t2;
		}
		float n2;
		int n;
		s1r_data[tid] = 0; s1i_data[tid] = 0; s2r_data[tid] = 0; s2i_data[tid] = 0;
		for(n = 1; n <= nmax; n++)
		{
			n2 = (2 * n + 1.0) / (n * (n + 1));
			s1r_data[tid] = s1r_data[tid] + (abr_data[0*nmax + n - 1] * p[n - 1 + tid * nmax] + abr_data[1*nmax+n-1] * t[n-1 + tid * nmax])*n2;
			s1i_data[tid] = s1i_data[tid] + (abi_data[0*nmax + n - 1] * p[n - 1 + tid * nmax] + abi_data[1*nmax+n-1] * t[n-1 + tid * nmax])*n2;
	
			s2r_data[tid] = s2r_data[tid] + (abr_data[0*nmax + n - 1] * t[n - 1 + tid * nmax] + abr_data[1*nmax+n-1] * p[n-1 + tid * nmax])*n2;
			s2i_data[tid] = s2i_data[tid] + (abi_data[0*nmax + n - 1] * t[n - 1 + tid * nmax] + abi_data[1*nmax+n-1] * p[n-1 + tid * nmax])*n2;
		}
	}
	
}


//extern "C" int GPUInitialization()
//{
//	int devID = findCudaDevice(0, NULL);
//	return devID;
//	float mr, mi;				//real and image components of m
//	float x;
//	int N = 10001;
//	float *u = (float*)malloc(N*sizeof(float));
//	int nmax = 665;
//	float* abr = (float*)malloc(4*nmax*sizeof(float));
//	float* abi = (float*)malloc(4*nmax*sizeof(float));			//real and image components of ab
//	float *s1r, *s1i, *s2r, *s2i;	
//	s1r = (float*)malloc(N*sizeof(float));
//	s1i = (float*)malloc(N*sizeof(float));
//	
//	s2r = (float*)malloc(N*sizeof(float));
//	s2i = (float*)malloc(N*sizeof(float));
//	for(int i = 0; i < N; i++)
//	{
//		s2r[i] = i;
//		s2i[i] = -i;
//	}
//
//	const unsigned int num_threads = N;
//	 
//	float *u_data;
//	float *abr_data;
//	float *abi_data;
//	float *s1r_data;
//	float *s1i_data;
//	float *s2r_data;
//	float *s2i_data;
//	float *p_data;
//	float *t_data; 
//	FILE* file;
//	file = fopen("D:\\ff.txt", "r");
//	float a0, a1, a2, a3;
//	for(int i = 0; i < nmax; i++)
//	{
//		fscanf(file, "%lf %lf %lf %lf", &a0, &a1, &a2, &a3);
//		abr[i] = a0;
//		abr[nmax + i] = a2;
//		abi[i] = a1;
//		abi[nmax+i] = a3;
//	}
//
//	fclose(file);
//	mr = 1.5; mi = 0; x = 628.31853; 
//	for(int i =0; i < N; i++)
//		u[i] = cos(3.1415927/(N-1)*i);
//    checkCudaErrors(hipMalloc((void **) &u_data, N*sizeof(float)));
//    checkCudaErrors(hipMalloc((void **) &abr_data, 4*nmax*sizeof(float)));
//    checkCudaErrors(hipMalloc((void **) &abi_data, 4*nmax*sizeof(float)));
//    checkCudaErrors(hipMalloc((void **) &s1r_data, N*sizeof(float)));
//    checkCudaErrors(hipMalloc((void **) &s1i_data, N*sizeof(float)));
//    checkCudaErrors(hipMalloc((void **) &s2r_data, N*sizeof(float)));
//    checkCudaErrors(hipMalloc((void **) &s2i_data, N*sizeof(float)));
//    checkCudaErrors(hipMalloc((void **) &p_data, N*nmax*sizeof(float)));
//    checkCudaErrors(hipMalloc((void **) &t_data, N*nmax*sizeof(float)));
//    // copy host memory to device
//    checkCudaErrors(hipMemcpy(u_data, u, N*sizeof(float), hipMemcpyHostToDevice));
//
//	hipEvent_t start, stop;  
//	float time;  
//	hipEventCreate(&start);  
//	hipEventCreate(&stop); 
//	
//	hipEventRecord(start, 0);
//
//    checkCudaErrors(hipMemcpy(abr_data, abr, 4*nmax*sizeof(float), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpy(abi_data, abi, 4*nmax*sizeof(float), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpy(s2r_data, s2r, N*sizeof(float), hipMemcpyHostToDevice));
//    checkCudaErrors(hipMemcpy(s2i_data, s2i, N*sizeof(float), hipMemcpyHostToDevice));
//
//    dim3 grid(N/192+1, 1, 1);
//    dim3 block(12, 16, 1);
//
//	//for(int i = 0; i < 20; i++)
//		kernel<<< grid, block >>>(mr, mi, x, (float *) u_data, N, nmax, abr_data, abi_data,s1r_data, s1i_data, s2r_data, s2i_data, p_data, t_data);
//
//	hipEventRecord(stop, 0);  
//	hipEventSynchronize(stop);  
//  
//	hipEventElapsedTime(&time, start, stop);  
//	hipEventDestroy(start);  
//	hipEventDestroy(stop);
//
//	
//    checkCudaErrors(hipMemcpy(u, u_data, N*sizeof(float),hipMemcpyDeviceToHost));
//
//
//
//
//    getLastCudaError("Kernel execution failed");
//	
//    checkCudaErrors(hipMemcpy(s1r, s1r_data, N*sizeof(float), hipMemcpyDeviceToHost));
//    checkCudaErrors(hipMemcpy(s1i, s1i_data, N*sizeof(float), hipMemcpyDeviceToHost));
//    checkCudaErrors(hipMemcpy(s2r, s2r_data, N*sizeof(float), hipMemcpyDeviceToHost));
//    checkCudaErrors(hipMemcpy(s2i, s2i_data, N*sizeof(float), hipMemcpyDeviceToHost));
//	
//	checkCudaErrors(hipFree(s1r_data));
//	checkCudaErrors(hipFree(s1i_data));
//	checkCudaErrors(hipFree(s2r_data));
//	checkCudaErrors(hipFree(s2i_data));
//	checkCudaErrors(hipFree(u_data));
//	checkCudaErrors(hipFree(abr_data));
//	checkCudaErrors(hipFree(abi_data));
//
//	free(u);
//    free(abr);
//    free(abi);
//    free(s1r);
//    free(s1i);
//    free(s2r);
//    free(s2i);
//	return true;
//}
//


extern "C" int Mie_S12()
{
	int devID = findCudaDevice(0, NULL);
	float mr, mi;				//real and image components of m
	float x;
	int N = 10001;
	float *u = (float*)malloc(N*sizeof(float));
	int nmax = 665;
	float* abr = (float*)malloc(4*nmax*sizeof(float));
	float* abi = (float*)malloc(4*nmax*sizeof(float));			//real and image components of ab
	float *s1r, *s1i, *s2r, *s2i;	
	s1r = (float*)malloc(N*sizeof(float));
	s1i = (float*)malloc(N*sizeof(float));
	
	s2r = (float*)malloc(N*sizeof(float));
	s2i = (float*)malloc(N*sizeof(float));
	for(int i = 0; i < N; i++)
	{
		s2r[i] = i;
		s2i[i] = -i;
	}

	const unsigned int num_threads = N;
	 
	float *u_data;
	float *abr_data;
	float *abi_data;
	float *s1r_data;
	float *s1i_data;
	float *s2r_data;
	float *s2i_data;
	float *p_data;
	float *t_data; 
	FILE* file;
	file = fopen("D:\\ff.txt", "r");
	float a0, a1, a2, a3;
	for(int i = 0; i < nmax; i++)
	{
		fscanf(file, "%lf %lf %lf %lf", &a0, &a1, &a2, &a3);
		abr[i] = a0;
		abr[nmax + i] = a2;
		abi[i] = a1;
		abi[nmax+i] = a3;
	}

	fclose(file);
	mr = 1.5; mi = 0; x = 628.31853; 
	for(int i =0; i < N; i++)
		u[i] = cos(3.1415927/(N-1)*i);
    checkCudaErrors(hipMalloc((void **) &u_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &abr_data, 4*nmax*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &abi_data, 4*nmax*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &s1r_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &s1i_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &s2r_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &s2i_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &p_data, N*nmax*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &t_data, N*nmax*sizeof(float)));
    // copy host memory to device
    checkCudaErrors(hipMemcpy(u_data, u, N*sizeof(float), hipMemcpyHostToDevice));

	hipEvent_t start, stop;  
	float time;  
	hipEventCreate(&start);  
	hipEventCreate(&stop); 
	
	hipEventRecord(start, 0);

    checkCudaErrors(hipMemcpy(abr_data, abr, 4*nmax*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(abi_data, abi, 4*nmax*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(s2r_data, s2r, N*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(s2i_data, s2i, N*sizeof(float), hipMemcpyHostToDevice));

    dim3 grid(N/192+1, 1, 1);
    dim3 block(12, 16, 1);

	//for(int i = 0; i < 20; i++)
		kernel<<< grid, block >>>(mr, mi, x, (float *) u_data, N, nmax, abr_data, abi_data,s1r_data, s1i_data, s2r_data, s2i_data, p_data, t_data);

	hipEventRecord(stop, 0);  
	hipEventSynchronize(stop);  
  
	hipEventElapsedTime(&time, start, stop);  
	hipEventDestroy(start);  
	hipEventDestroy(stop);

	
    checkCudaErrors(hipMemcpy(u, u_data, N*sizeof(float),hipMemcpyDeviceToHost));




    getLastCudaError("Kernel execution failed");
	
    checkCudaErrors(hipMemcpy(s1r, s1r_data, N*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(s1i, s1i_data, N*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(s2r, s2r_data, N*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(s2i, s2i_data, N*sizeof(float), hipMemcpyDeviceToHost));
	
	checkCudaErrors(hipFree(s1r_data));
	checkCudaErrors(hipFree(s1i_data));
	checkCudaErrors(hipFree(s2r_data));
	checkCudaErrors(hipFree(s2i_data));
	checkCudaErrors(hipFree(u_data));
	checkCudaErrors(hipFree(abr_data));
	checkCudaErrors(hipFree(abi_data));

	free(u);
    free(abr);
    free(abi);
    free(s1r);
    free(s1i);
    free(s2r);
    free(s2i);
	return true;
}

BOOL _tmain(int argc, _TCHAR* argv[])
{
	if(Mie_S12() < 0)
		return false;
//    CPUMieS12(m, x, Math.Cos(calculationParameter.planewaveScatteringAngle[i]),nmax,ab);
	return 0;
}

