#include "hip/hip_runtime.h"
// GPUMieScattering.cpp : �������̨Ӧ�ó������ڵ㡣
//
#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>
#include <helper_functions.h>
#include <hip/device_functions.h>
#include <time.h>
#include "GPUMieScatteringDll.h"
#define PI 3.141592654



BOOL APIENTRY DllMain( HMODULE hModule,
                       DWORD  ul_reason_for_call,
                       LPVOID lpReserved
					 )
{
	switch (ul_reason_for_call)
	{
	case DLL_PROCESS_ATTACH:
	case DLL_THREAD_ATTACH:
	case DLL_THREAD_DETACH:
	case DLL_PROCESS_DETACH:
		break;
	}
	return TRUE;
}

__global__ void kernel(float mr, float mi, float x, float *u_data, int N, int nmax, float* abr_data, 
	float* abi_data,float* s1r_data, float* s1i_data, float* s2r_data, float* s2i_data, float* p, float *t)
{

    unsigned int tid = threadIdx.x + threadIdx.y * blockDim.x + blockIdx.x * blockDim.x * blockDim.y;
    float u = u_data[tid];
	if(tid < N)
	{
		p[0 + tid * nmax] = 1; t[0 + tid * nmax] = u;
		p[1 + tid * nmax] = 3 * u; t[1 + tid * nmax] = 3 * cos(2 * acos(u));
		float p1, p2, t1, t2;
		int n1;
		for (n1 = 3; n1 <= nmax; n1++)
		{
			p1 = (2 * n1 - 1.0) / (n1 - 1) * p[n1 - 2 + tid * nmax] * u;
			p2 = n1 * 1.0 / (n1 - 1) * p[n1 - 3 + tid * nmax];
			p[n1 - 1 + tid * nmax] = p1 - p2;
			t1 = n1 * u * p[n1 - 1 + tid * nmax];
			t2 = (n1 + 1) * p[n1 - 2 + tid * nmax];
			t[n1 - 1 + tid * nmax] = t1 - t2;
		}
		float n2;
		int n;
		s1r_data[tid] = 0; s1i_data[tid] = 0; s2r_data[tid] = 0; s2i_data[tid] = 0;
		for(n = 1; n <= nmax; n++)
		{
			n2 = (2 * n + 1.0) / (n * (n + 1));
			s1r_data[tid] = s1r_data[tid] + (abr_data[0*nmax + n - 1] * p[n - 1 + tid * nmax] + abr_data[1*nmax+n-1] * t[n-1 + tid * nmax])*n2;
			s1i_data[tid] = s1i_data[tid] + (abi_data[0*nmax + n - 1] * p[n - 1 + tid * nmax] + abi_data[1*nmax+n-1] * t[n-1 + tid * nmax])*n2;
	
			s2r_data[tid] = s2r_data[tid] + (abr_data[0*nmax + n - 1] * t[n - 1 + tid * nmax] + abr_data[1*nmax+n-1] * p[n-1 + tid * nmax])*n2;
			s2i_data[tid] = s2i_data[tid] + (abi_data[0*nmax + n - 1] * t[n - 1 + tid * nmax] + abi_data[1*nmax+n-1] * p[n-1 + tid * nmax])*n2;
		}
	}
	
}
//__global__ void ScatteringSuperposition(int integrationStepNumber, int lightsheetScatteringAngleNumber, float* lightsheetScatteringAngle_data, float*  lightsheetScatteringAmplitudeReal_data, float* lightsheetScatteringAmplitudeImage_data,
//		int planewaveScatteringAngleNumber,  float* planewaveScatteringAngle_data,  float* planewaveScatteringAmplitudeReal_data, float* planewaveScatteringAmplitudeImage_data,
//		int spectrumSampleNumber,  float* planewaveSpectrumAngle_data,  float* planewaveSpectrumReal_data, float* planewaveSpectrumImage_data)
//{
//
//}

float *u_data;
float *abr_data;
float *abi_data;
float *s1r_data;
float *s1i_data;
float *s2r_data;
float *s2i_data;
float *p_data;
float *t_data;
//float* lightsheetScatteringAngle_data;
//float *lightsheetScatteringAmplitudeReal_data;
//float *lightsheetScatteringAmplitudeImage_data; // results of light sheet scattering								//
//float* planewaveScatteringAngle_data;
//float *planewaveScatteringAmplitudeReal_data;
//float *planewaveScatteringAmplitudeImage_data;  //results of plane wave scattering
//float* planewaveSpectrumAngle_data;
//float *planewaveSpectrumReal_data;
//float *planewaveSpectrumImage_data;


hipEvent_t start, stop;  
float processingTime;
extern "C" _declspec( dllexport ) int Mie_S12(float mr, float mi, float x, int N, int nmax, float* s1r, float *s1i, float *s2r, float *s2i)
{
	//FILE* f;
	//f = fopen("D:\\ff.txt", "w+");
	const unsigned int num_threads = N;
	hipEventRecord(start, 0);
	
    dim3 grid(N/192+1, 1, 1);
    dim3 block(12, 16, 1);
	//fprintf(f,"gridx %d\n", N/192+1); 
	kernel<<< grid, block >>>(mr, mi, x, (float *) u_data, N, nmax, abr_data, abi_data,s1r_data, s1i_data, s2r_data, s2i_data, p_data, t_data);
	
	//fprintf(f,"After kernel\n");
	hipEventRecord(stop, 0);  
	hipEventSynchronize(stop);  
  
	hipEventElapsedTime(&processingTime, start, stop);  

	
	//fprintf(f,"processingTime %f\n", processingTime);
	checkCudaErrors(hipMemcpy(s1r, s1r_data, N*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(s1i, s1i_data, N*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(s2r, s2r_data, N*sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(s2i, s2i_data, N*sizeof(float), hipMemcpyDeviceToHost));
	//fclose(f);
	return true;
}




extern "C" _declspec( dllexport ) int GPUDeInitialization()
{
	hipEventDestroy(start);  
	hipEventDestroy(stop);
	checkCudaErrors(hipFree(s1r_data));
	checkCudaErrors(hipFree(s1i_data));
	checkCudaErrors(hipFree(s2r_data));
	checkCudaErrors(hipFree(s2i_data));
	checkCudaErrors(hipFree(u_data));
	checkCudaErrors(hipFree(abr_data));
	checkCudaErrors(hipFree(abi_data));
	checkCudaErrors(hipFree(p_data));
	checkCudaErrors(hipFree(t_data));

	//checkCudaErrors(hipFree(lightsheetScatteringAngle_data));
 //   checkCudaErrors(hipFree(lightsheetScatteringAmplitudeReal_data));
 //   checkCudaErrors(hipFree(lightsheetScatteringAmplitudeImage_data));
 //   checkCudaErrors(hipFree(planewaveScatteringAngle_data));
 //   checkCudaErrors(hipFree(planewaveScatteringAmplitudeReal_data));
 //   checkCudaErrors(hipFree(planewaveScatteringAmplitudeImage_data));
 //   checkCudaErrors(hipFree(planewaveSpectrumAngle_data));
 //   checkCudaErrors(hipFree(planewaveSpectrumReal_data));
 //   checkCudaErrors(hipFree(planewaveSpectrumImage_data));
    
	
	
	return true;
}



extern "C" _declspec( dllexport ) int GPUInitialization(int N, int nmax, float *u, float *abr, float *abi)
{
	int devID = -5;
	devID = findCudaDevice(0, NULL);
	const unsigned int num_threads = N;
    checkCudaErrors(hipMalloc((void **) &u_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &abr_data, 4*nmax*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &abi_data, 4*nmax*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &s1r_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &s1i_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &s2r_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &s2i_data, N*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &p_data, N*nmax*sizeof(float)));
    checkCudaErrors(hipMalloc((void **) &t_data, N*nmax*sizeof(float)));
    


	//checkCudaErrors(hipMalloc((void **) &lightsheetScatteringAngle_data, 100000*sizeof(float)));
 //   checkCudaErrors(hipMalloc((void **) &lightsheetScatteringAmplitudeReal_data, 100000*sizeof(float)));
 //   checkCudaErrors(hipMalloc((void **) &lightsheetScatteringAmplitudeImage_data, 100000*sizeof(float)));
 //   checkCudaErrors(hipMalloc((void **) &planewaveScatteringAngle_data, 100000*sizeof(float)));
 //   checkCudaErrors(hipMalloc((void **) &planewaveScatteringAmplitudeReal_data, 100000*sizeof(float)));
 //   checkCudaErrors(hipMalloc((void **) &planewaveScatteringAmplitudeImage_data, 100000*sizeof(float)));
 //   checkCudaErrors(hipMalloc((void **) &planewaveSpectrumReal_data, 100000*sizeof(float)));
 //   checkCudaErrors(hipMalloc((void **) &planewaveSpectrumImage_data, 100000*sizeof(float)));
 //   checkCudaErrors(hipMalloc((void **) &planewaveSpectrumAngle_data, 100000*sizeof(float)));
    
	

	checkCudaErrors(hipMemcpy(u_data, u, N*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(abr_data, abr, 4*nmax*sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(abi_data, abi, 4*nmax*sizeof(float), hipMemcpyHostToDevice));
	
	float time;  
	hipEventCreate(&start);  
	hipEventCreate(&stop); 
	return devID;
}

//extern "C" int MieScatteringSuperposition(int integrationStepNumber, 
//	int lightsheetScatteringAngleNumber, float* lightsheetScatteringAngle, float *lightsheetScatteringAmplitudeReal, float *lightsheetScatteringAmplitudeImage, // results of light sheet scattering								//
//	int planewaveScatteringAngleNumber, float* planewaveScatteringAngle, float *planewaveScatteringAmplitudeReal, float *planewaveScatteringAmplitudeImage,  //results of plane wave scattering
//	int spectrumSampleNumber, float* planewaveSpectrumAngle, float *planewaveSpectrumReal, float *planewaveSpectrumImage)			//plane wave spectrum of a light sheet
//{
//	const unsigned int num_threads = lightsheetScatteringAngleNumber;
//	hipEventRecord(start, 0);
//
//
//	checkCudaErrors(hipMemcpy(lightsheetScatteringAngle_data, lightsheetScatteringAngle, lightsheetScatteringAngleNumber*sizeof(float), hipMemcpyHostToDevice));
//
//	checkCudaErrors(hipMemcpy(planewaveScatteringAmplitudeReal_data, planewaveScatteringAmplitudeReal, planewaveScatteringAngleNumber*sizeof(float), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(planewaveScatteringAmplitudeImage_data, planewaveScatteringAmplitudeImage, planewaveScatteringAngleNumber*sizeof(float), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(planewaveScatteringAngle_data, planewaveScatteringAngle, planewaveScatteringAngleNumber*sizeof(float), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(planewaveSpectrumReal_data, planewaveSpectrumReal, spectrumSampleNumber*sizeof(float), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(planewaveSpectrumImage_data, planewaveSpectrumImage, spectrumSampleNumber*sizeof(float), hipMemcpyHostToDevice));
//	checkCudaErrors(hipMemcpy(planewaveSpectrumAngle_data, planewaveSpectrumAngle, spectrumSampleNumber*sizeof(float), hipMemcpyHostToDevice));
//	
//
//
//
//    dim3 grid(num_threads/192+1, 1, 1);
//    dim3 block(12, 16, 1);
//	//fprintf(f,"gridx %d\n", N/192+1); 
//	ScatteringSuperposition<<< grid, block >>>(integrationStepNumber, lightsheetScatteringAngleNumber, lightsheetScatteringAngle_data, lightsheetScatteringAmplitudeReal_data, lightsheetScatteringAmplitudeImage_data,
//		planewaveScatteringAngleNumber, planewaveScatteringAngle_data, planewaveScatteringAmplitudeReal_data, planewaveScatteringAmplitudeImage_data,
//		spectrumSampleNumber, planewaveSpectrumAngle_data, planewaveSpectrumReal_data, planewaveSpectrumImage_data);
//	
//	//fprintf(f,"After kernel\n");
//	hipEventRecord(stop, 0);  
//	hipEventSynchronize(stop);  
//  
//	hipEventElapsedTime(&processingTime, start, stop);  
//
//	
//	//fprintf(f,"processingTime %f\n", processingTime);
//	checkCudaErrors(hipMemcpy(lightsheetScatteringAmplitudeReal, lightsheetScatteringAmplitudeReal_data, lightsheetScatteringAngleNumber*sizeof(float), hipMemcpyDeviceToHost));
//	checkCudaErrors(hipMemcpy(lightsheetScatteringAmplitudeImage, lightsheetScatteringAmplitudeImage_data, lightsheetScatteringAngleNumber*sizeof(float), hipMemcpyDeviceToHost));
//
//	//fclose(f);
//	return true;
//	
//}

//extern "C" _declspec( dllexport ) int fadd2add(int a, int b)
//{
//	return a+b;
//}
extern "C" _declspec( dllexport ) float GetProcessingTime()
{
	return processingTime;
}